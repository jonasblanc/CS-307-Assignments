/*
============================================================================
Filename    : implementation.cu
Author      : Jonas Blanc, Mélissa Gehring
SCIPER      : 287508, 264265
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define INPUT(I,J) input[(I)*length+(J)]
#define S_DATA(I,J) sdata[(I)*s_length+(J)]

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__ void compute_one_iteration(double *input, double *output, int length) {
	int j = (blockIdx.x * blockDim.x) + threadIdx.x ; 
    int i = (blockIdx.y * blockDim.y) + threadIdx.y ;
    int array_index = (i * length) + j;

	if(0 < i && i < length-1 && 0 < j && j < length - 1){
		output[array_index] = (input[(i-1)*(length)+(j-1)] +
                                input[(i-1)*(length)+(j)]   +
                                input[(i-1)*(length)+(j+1)] +
                                input[(i)*(length)+(j-1)]   +
                                input[(i)*(length)+(j)]     +
                                input[(i)*(length)+(j+1)]   +
                                input[(i+1)*(length)+(j-1)] +
                                input[(i+1)*(length)+(j)]   +
                                input[(i+1)*(length)+(j+1)] ) / 9;
	}
	output[(length/2-1)*length+(length/2-1)] = 1000;
    output[(length/2)*length+(length/2-1)]   = 1000;
    output[(length/2-1)*length+(length/2)]   = 1000;
    output[(length/2)*length+(length/2)]     = 1000;
 }

 __global__ void compute_one_iteration_smart_mid(double *input, double *output, int length) {

    int j = (blockIdx.x * blockDim.x) + threadIdx.x ; 
    int i = (blockIdx.y * blockDim.y) + threadIdx.y ;
    int array_index = (i * length) + j;

    int mid1 = (length/2-1)*length+(length/2-1);
    int mid2 = (length/2)*length+(length/2-1);
    int mid3 = (length/2-1)*length+(length/2);
    int mid4 = (length/2)*length+(length/2);

    if(array_index == mid1 || array_index == mid2 || array_index == mid3 || array_index == mid4){
        return;
    }

	if(0 < i && i < length-1 && 0 < j && j < length - 1){
		output[array_index] = (input[(i-1)*(length)+(j-1)] +
                                input[(i-1)*(length)+(j)]   +
                                input[(i-1)*(length)+(j+1)] +
                                input[(i)*(length)+(j-1)]   +
                                input[(i)*(length)+(j)]     +
                                input[(i)*(length)+(j+1)]   +
                                input[(i+1)*(length)+(j-1)] +
                                input[(i+1)*(length)+(j)]   +
                                input[(i+1)*(length)+(j+1)] ) / 9;
    }
 }

 __global__ void compute_one_iteration_shared(double *input, double *output, int length) {
    extern __shared__ double sdata[]; // Used in macro S_DATA
    
    int j = (blockIdx.x * (blockDim.x - 2)) + threadIdx.x ; 
    int i = (blockIdx.y * (blockDim.y - 2)) + threadIdx.y ;
    int array_index = (i * length) + j;

    int s_i = threadIdx.y;
    int s_j = threadIdx.x;
    int s_length = blockDim.x; // Used in macro S_DATA

    // Load shared memory
    if(0 <= i && i <= length-1 && 0 <= j && j <= length - 1){
        S_DATA(s_i, s_j) = INPUT(i,j);
        __syncthreads();
    }

    if(0 < s_i && s_i < s_length-1 && 0 < s_j && s_j < s_length - 1){
        if(0 < i && i < length-1 && 0 < j && j < length - 1){
            output[array_index] = ( S_DATA(s_i - 1, s_j -1 )    +
                                    S_DATA(s_i - 1, s_j)        +
                                    S_DATA(s_i - 1, s_j + 1)    +
                                    S_DATA(s_i, s_j - 1)        +
                                    S_DATA(s_i, s_j)            +
                                    S_DATA(s_i, s_j + 1)        +
                                    S_DATA(s_i + 1, s_j - 1)    +
                                    S_DATA(s_i + 1, s_j)        +
                                    S_DATA(s_i + 1, s_j + 1)    ) / 9;
        }
    }

	output[(length/2-1)*length+(length/2-1)] = 1000;
    output[(length/2)*length+(length/2-1)]   = 1000;
    output[(length/2-1)*length+(length/2)]   = 1000;
    output[(length/2)*length+(length/2)]     = 1000;
 }


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing */
    size_t array_size = length * length * sizeof(double);
    double* gpu_array_in;
    hipMalloc( (void**)&gpu_array_in, array_size);
    double* gpu_array_out;
    hipMalloc( (void**)&gpu_array_out, array_size);

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device */
    hipMemcpy((void*)gpu_array_in, (void*)input, array_size, hipMemcpyHostToDevice);
    hipMemcpy((void*)gpu_array_out, (void*)output, array_size, hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    hipEventRecord(comp_start);
    /* GPU calculation */
    // SQUARE THREADS BLOCKS
    size_t threadBlockSide = 8;
    size_t nbBlockSide = length / threadBlockSide;

    // If not a multiple
    if(length % threadBlockSide != 0){
        nbBlockSide += 1;
    }

    dim3 thrsPerBlock(threadBlockSide,threadBlockSide); 
    dim3 nBlks(nbBlockSide,nbBlockSide);   
    
    // SHARED MEMORY
    size_t threadBlockSide_shared = 32;
    size_t nbBlockSide_shared = length / (threadBlockSide_shared -2);

    // If not a multiple
    if(length % (threadBlockSide_shared -2) != 0){
        nbBlockSide_shared += 1;
    }

    size_t smemSize_shared = threadBlockSide_shared * threadBlockSide_shared * sizeof(double);

    dim3 thrsPerBlock_shared(threadBlockSide_shared, threadBlockSide_shared); 
    dim3 nBlks_shared(nbBlockSide_shared, nbBlockSide_shared); 

    // ROW THREADS BLOCKS
    size_t threadBlockSide_row = length;
    size_t nbBlockSide_row = 1;

    if(threadBlockSide_row > 1024){
        threadBlockSide_row = 512;
        nbBlockSide_row = length / threadBlockSide_row;

        // If not a multiple
        if(length % threadBlockSide_row != 0){
            nbBlockSide_row += 1;
        }
    }

    dim3 thrsPerBlock_row(threadBlockSide_row, 1); 
    dim3 nBlks_row(nbBlockSide_row, length); 
    
    double *temp;
    for(int n = 0; n < iterations; n++)
    {
        //compute_one_iteration <<< nBlks, thrsPerBlock >>> (gpu_array_in, gpu_array_out, length);
        compute_one_iteration_smart_mid <<< nBlks, thrsPerBlock >>> (gpu_array_in, gpu_array_out, length);
        //compute_one_iteration <<< nBlks_row, thrsPerBlock_row >>> (gpu_array_in, gpu_array_out, length);
        //compute_one_iteration_shared <<< nBlks_shared, thrsPerBlock_shared, smemSize_shared >>> (gpu_array_in, gpu_array_out, length);

        temp = gpu_array_in;
        gpu_array_in = gpu_array_out;
        gpu_array_out = temp;
    }
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output, (void*)gpu_array_in, array_size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree((void*)gpu_array_in);
    hipFree((void*)gpu_array_out);
    
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
